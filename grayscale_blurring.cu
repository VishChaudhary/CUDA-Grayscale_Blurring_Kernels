#include "hip/hip_runtime.h"
#include "stdio.h"
#include "math.h"

#define N 3
#define BLUR_SIZE (N-1/2)
#define channels 3

__global__  void color2grayscale(unsigned char *Pin, unsigned char *Pout, int width, int height){
  int row_num = blockIdx.y * blockDim.y + threadIdx.y;
  int col_num = blockIdx.x * blockDim.x + threadIdx.x;

  if(row_num < height && col_num < width){
    int grayscaleOffset = row_num * width + col_num;

    int rgbOffset = grayscaleOffset*channels;

    unsigned char r = Pin[rgbOffset];
    unsigned char g = Pin[rgbOffset + 1];
    unsigned char b = Pin[rgbOffset + 2];

    Pout[grayscaleOffset] = 0.21*r + 0.71*g + 0.07*b;
  }
}

__global__ void blurKernel(unsigned char* Pin, unsigned char* Pout, int width, int height){
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  int row = blockIdx.y * blockDim.y + threadIdx.y;

  if(col < width && row< height){
    int pixelVal = 0;
    int pixelCount = 0;

    for(int blurRow = -BLUR_SIZE; blurRow < BLUR_SIZE + 1; ++blurRow){
      for(int blurCol = -BLUR_SIZE; blurCol < BlUR_SIZE + 1; ++blurCol){
          int currRow = row + blurRow;
          int currCol = col + blurCol;

          if(currRow>= 0 && currRow < height && currCol >= 0 && currCol < width){
            pixelVal += Pin[currRow*width+currCol];
            pixelCount++;
          }
      }
    }
    Pout[row*width+col] = (unsigned char) (pixelVal/pixelCount++;);
  }
}

int main (){
  // Load in some image into 
  // Get the vertical (n) and horizontal (m) dimmensions of the image
  // Linearize the 2D image into a 1D array (Pin_h)
  // Create Pin_d and allocate its memory (hipMalloc)
  // hipMemcpy(Pin_d, Pin_h, size, hipMemcpyHostToDevice);

  // Pin_h
  // Pout_h
  // Pin_d
  // Pout_d

  
  // Create a pointer to Pout_d allocate its memory (hipMalloc) 
  // hipMemcpy(Pout_d, Pout_h, size, hipMemcpyHostToDevice);
  dim3 dimGrid(ceil(m/16.0), ceil(n/16.0), 1);
  dim3 dimBlock(16,16,1);

  color2grayscale<<<dimGrid, dimBlock>>> (Pin_d, Pout_d, m , n);
  // hipMemcpy(Pout_h, Pout_d, size, hipMemcpyDeviceToDevice);
  
}
