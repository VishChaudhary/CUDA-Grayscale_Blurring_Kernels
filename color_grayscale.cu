#include "hip/hip_runtime.h"
#include "stdio.h"
#include "math.h"

#define channels 3

__global__  void color2grayscale(unsigned char *Pin, unsigned char *Pout, int width, int height){
  int row_num = blockIdx.y * blockDim.y + threadIdx.y;
  int col_num = blockIdx.x * blockDim.x + threadIdx.x;

  if(row_num < height && col_num < width){
    int grayscaleOffset = row_num * width + col_num;

    int rgbOffset = grayscaleOffset*channels;

    unsigned char r = Pin[rgbOffset];
    unsigned char g = Pin[rgbOffset + 1];
    unsigned char b = Pin[rgbOffset + 2];

    Pout[grayscaleOffset] = 0.21*r + 0.71*g + 0.07*b;
  }
}

int main (){
  // Load in some image into 
  // Get the vertical (n) and horizontal (m) dimmensions of the image
  // Linearize the 2D image into a 1D array (Pin_h)
  // Create Pin_d and allocate its memory (hipMalloc)
  // hipMemcpy(Pin_d, Pin_h, size, hipMemcpyHostToDevice);

  // Pin_h
  // Pout_h
  // Pin_d
  // Pout_d

  
  // Create a pointer to Pout_d allocate its memory (hipMalloc) 
  // hipMemcpy(Pout_d, Pout_h, size, hipMemcpyHostToDevice);
  dim3 dimGrid(ceil(m/16.0), ceil(n/16.0), 1);
  dim3 dimBlock(16,16,1);

  color2grayscale<<<dimGrid, dimBlock>>> (Pin_d, Pout_d, m , n);
  // hipMemcpy(Pout_h, Pout_d, size, hipMemcpyDeviceToDevice);
  
}
